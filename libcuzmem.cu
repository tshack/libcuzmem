#include "hip/hip_runtime.h"
/*  This file is part of libcuzmem
    Copyright (C) 2011  James A. Shackleford

    libcuzmem is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "libcuzmem.h"

#define DEBUG

//------------------------------------------------------------------------------
// STATE SYMBOLS                            ...I know!
//------------------------------------------------------------------------------
char plan_name[FILENAME_MAX];
char project_name[FILENAME_MAX];
int (*call_tuner)(enum cuzmem_tuner_action) = cuzmem_tuner_exhaustive;
unsigned int current_knob = 0;
unsigned int num_knobs = 0;
unsigned int tune_iter = 0;
unsigned int tune_iter_max = 0;
enum cuzmem_op_mode op_mode = CUZMEM_RUN;


//------------------------------------------------------------------------------
// CUDA RUNTIME REPLACEMENTS
//------------------------------------------------------------------------------

hipError_t
hipMalloc (void **devPtr, size_t size)
{
    int use_global;
    hipError_t driver_return;
    hipDeviceptr_t dev_mem;
    void* host_mem;

    // Decide what to do with current knob
    if (CUZMEM_RUN == op_mode) {
        // 1) Load plan for this project
//        plan_load (plan_name, project_name);

        // 2) Using current_knob, determine malloc location
        use_global = call_tuner (CUZMEM_TUNER_LOOKUP);

        // 3) Allocate either pinned host or global device memory
    }
    else if (CUZMEM_TUNE == op_mode) {
        // 1) Load plan draft for this iteration

        // 2) Lookup current_knob in plan draft, determine malloc location
        use_global = call_tuner (CUZMEM_TUNER_LOOKUP);

        // 3) Allocate either pinned host or global device memory
    }

    if (use_global) {
        // allocate gpu global memory
        driver_return = hipMalloc (&dev_mem, (unsigned int)size);
    } else {
        // allocate pinned host memory (probably broken for now)
        driver_return = hipMemAllocHost ((void **)&host_mem, (unsigned int)size);
        if (driver_return != hipSuccess) { return hipErrorOutOfMemory; };
        driver_return = hipHostGetDevicePointer (&dev_mem, host_mem, 0);
    }
    *devPtr = (void *)dev_mem;

#if defined (DEBUG)
    printf ("libcuzmem: %s:%s | %i KB  [%i/%i] ondev:%i\n",
            project_name, plan_name, (unsigned int)(size / 1024),
            current_knob, num_knobs-1, use_global);
#endif

    // Get ready for next knob
    current_knob++;

    // Morph CUDA Driver return codes into CUDA Runtime codes
    switch (driver_return)
    {
    case hipSuccess:
        return (hipSuccess);
    case hipErrorDeinitialized:
    case hipErrorNotInitialized:
    case hipErrorInvalidContext:
    case hipErrorInvalidValue:
    case hipErrorOutOfMemory:
    default:
        return (hipErrorOutOfMemory);
    }


}



//------------------------------------------------------------------------------
// FRAMEWORK FUNCTIONS
//------------------------------------------------------------------------------

// Called at start of each plan invocation
void
cuzmem_start (enum cuzmem_op_mode m)
{
    char debug_mode[20];

    // This state info is modified for all engines.
    current_knob = 0;
    op_mode = m;

#if defined (DEBUG)
    if (CUZMEM_RUN == op_mode) {
        strcpy (debug_mode, "CUZMEM_RUN");
    }
    else if (CUZMEM_TUNE == op_mode) {
        strcpy (debug_mode, "CUZMEM_TUNE");
    }
    else {
        printf ("libcuzmem: unknown operation mode specified! (exiting)\n");
        exit (1);
    }

    printf ("libcuzmem: mode is %s\n", debug_mode);
#endif

    // Invoke Tuner's "Start of Plan" routine.
    call_tuner (CUZMEM_TUNER_START);
}


// Called at end of each plan invocation
cuzmem_op_mode
cuzmem_end ()
{
    // Ask the selected Tuner Engine what to do.
    call_tuner (CUZMEM_TUNER_END);

    // Return this back to calling program so that the
    // framework will know what to do next: next iteration
    // or stop iterating.
    return op_mode;
}



//------------------------------------------------------------------------------
// USER INTERFACE FUNCTIONS
//------------------------------------------------------------------------------

// Used to select/define CUZMEM project
void
cuzmem_set_project (char* project)
{
    strcpy (project_name, project);
}


// Used to select/define CUZMEM plan
void
cuzmem_set_plan (char* plan)
{
    strcpy (plan_name, plan);
}


// Used to select Tuning Engine
void
cuzmem_set_tuner (enum cuzmem_tuner t)
{
    switch (t)
    {
    case CUZMEM_EXHAUSTIVE:
    default:
        call_tuner = cuzmem_tuner_exhaustive;
    }
}



//------------------------------------------------------------------------------
// TUNING ENGINES
//------------------------------------------------------------------------------

// The default Tuning Engine
int
cuzmem_tuner_exhaustive (enum cuzmem_tuner_action action)
{
    if (CUZMEM_RUN == op_mode) {
        if (CUZMEM_TUNER_START == action) {
            // For now, do nothing special.
            return 0;
        }
        else if (CUZMEM_TUNER_LOOKUP == action) {
            // For now, just blindly put everything in device global
            return 1;
        }
        else if (CUZMEM_TUNER_END == action) {
            // For now, do nothing special.
            return 0;
        }
        else {
            printf ("libcuzmem: tuner asked to perform unknown run action!\n");
            exit (1);
        }
    }
    else if (CUZMEM_TUNE == op_mode) {
        if (CUZMEM_TUNER_START == action) {
            // For now, do nothing special.
            printf ("libcuzmem: iteration %i/%i\n", tune_iter, tune_iter_max);
            return 0;
        }
        else if (CUZMEM_TUNER_LOOKUP == action) {
            // For now, just blindly put everything in device global
            return 1;
        }
        else if (CUZMEM_TUNER_END == action) {
            // Record # of malloc encounters
            if (current_knob > num_knobs) {
                num_knobs = current_knob;
                tune_iter_max = (unsigned int)pow (2, num_knobs);
            }

            // Increment tune iteration count
            tune_iter++;

            // Have we exhausted the search space?
            if (tune_iter >= tune_iter_max) {
                // If so, stop iterating
                op_mode = CUZMEM_RUN;
            }
                return 0;
            }
        else {
            printf ("libcuzmem: tuner asked to perform unknown tune action!\n");
            exit (1);
        }
    }

    // We should never get to here
    exit (1);
    return 0;
}
