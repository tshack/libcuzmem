/*  This file is part of libcuzmem
    Copyright (C) 2011  James A. Shackleford

    libcuzmem is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "libcuzmem.h"

#define DEBUG

char plan_name[FILENAME_MAX];
char project_name[FILENAME_MAX];
int (*search)() = cuzmem_search_exhaustive;
unsigned int current_knob = 0;
unsigned int num_knobs = 0;
enum cuzmem_mode mode = CUZMEM_RUN;


hipError_t
hipMalloc (void **devPtr, size_t size)
{
    int use_global;
    hipError_t driver_return;
    hipDeviceptr_t dev_mem;
    void* host_mem;

    // decide what to do with this... knob
    use_global = search();

    if (use_global) {
        // allocate gpu global memory
        driver_return = hipMalloc (&dev_mem, (unsigned int)size);
    } else {
        // allocate pinned host memory (probably broken for now)
        driver_return = hipMemAllocHost ((void **)&host_mem, (unsigned int)size);
        if (driver_return != hipSuccess) { return hipErrorOutOfMemory; };
        driver_return = hipHostGetDevicePointer (&dev_mem, host_mem, 0);
    }
    *devPtr = (void *)dev_mem;

#if defined (DEBUG)
    printf ("*** | %s : %s | [%i KB] - #%i/%i\n",
            project_name, plan_name, (unsigned int)(size / 1024),
            current_knob, num_knobs-1);
#endif

    current_knob++;

    switch (driver_return)
    {
    case hipSuccess:
        return (hipSuccess);
    case hipErrorDeinitialized:
    case hipErrorNotInitialized:
    case hipErrorInvalidContext:
    case hipErrorInvalidValue:
    case hipErrorOutOfMemory:
    default:
        return (hipErrorOutOfMemory);
    }


}


int
cuzmem_search_exhaustive ()
{
#if defined (DEBUG)
    printf ("** Going with Global!\n");
#endif

    // use gpu global memory
    return 1;
}


void
cuzmem_search (enum cuzmem_search_mode mode)
{
    switch (mode)
    {
    case CUZMEM_EXHAUSTIVE:
    case CUZMEM_MAGIC:
    default:
        search = cuzmem_search_exhaustive;
    }
}


void
cuzmem_start (enum cuzmem_mode m)
{
    mode = m;
    current_knob = 0;
    printf ("Mode is: %i\n", mode);
}


void
cuzmem_end ()
{
    if (current_knob > num_knobs) {
        num_knobs = current_knob;
    }
}


void
cuzmem_plan (char* plan)
{
    strcpy (plan_name, plan);
}


void
cuzmem_project (char* project)
{
    strcpy (project_name, project);
}

#if defined (commentout)
void
cuzmem_plan (int count, ...)
{
    int i;
    va_list knobs;

    va_start (knobs, count);

    for (i=0; i<count; i++) {
        
    }

    va_end (knobs);
}
#endif
