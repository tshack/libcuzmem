#include "hip/hip_runtime.h"
/*  This file is part of libcuzmem
    Copyright (C) 2011  James A. Shackleford

    libcuzmem is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "libcuzmem.h"
#include "plans.h"

//#define DEBUG

hipError_t alloc_mem (cuzmem_plan* entry, size_t size);
cuzmem_plan* cuzmem_tuner_exhaustive (enum cuzmem_tuner_action action, void* parm);

//------------------------------------------------------------------------------
// STATE SYMBOLS                            ...I know!
//------------------------------------------------------------------------------
char plan_name[FILENAME_MAX];
char project_name[FILENAME_MAX];
cuzmem_plan* (*call_tuner)(enum cuzmem_tuner_action, void*) = cuzmem_tuner_exhaustive;
unsigned int current_knob = 0;
unsigned int num_knobs = 0;
unsigned int tune_iter = 0;
unsigned int tune_iter_max = 0;
enum cuzmem_op_mode op_mode = CUZMEM_RUN;

cuzmem_plan *plan = NULL;

//------------------------------------------------------------------------------
// CUDA RUNTIME REPLACEMENTS
//------------------------------------------------------------------------------

hipError_t
hipMalloc (void **devPtr, size_t size)
{
    hipError_t ret;
    cuzmem_plan *entry = NULL;

    *devPtr = NULL;

    // Decide what to do with current knob
    if (CUZMEM_RUN == op_mode) {
        // 1) Load plan for this project
        entry = plan;

        // 2) Lookup malloc type for this knob & allocate
        while (entry != NULL) {
            if (entry->id == current_knob) {
                ret = alloc_mem (entry, size);
                *devPtr = entry->gpu_pointer;
                break;
            }
            entry = entry->next;
        }

        // Knob id exceeds those found in plan... must be in a malloc/free loop
        if (*devPtr == NULL) {
            // Look for a free()ed "inloop" marked plan entry 
            entry = plan;
            while (1) {
                if (entry == NULL) {
                    fprintf (stderr,"libcuzmem: unable to deduce inloop allocation from plan!\n");
                    exit (1);
                }
                if ((entry->inloop == 1)         &&
                    (entry->gpu_pointer == NULL) &&
                    (entry->size == size))
                {
                        ret = alloc_mem (entry, size);
                        if (ret != hipSuccess) {
                            fprintf (stderr, "libcuzmem: inloop alloc_mem() failed [%i]\n", ret);
                        }
                        *devPtr = entry->gpu_pointer;
                        break;
                }
                entry = entry->next;
            }
        } else {
            // Don't increment current_knob for inloop allocations,
            // they are knobs that we have already counted!
            current_knob++;
        }
    }
    else if (CUZMEM_TUNE == op_mode) {
        // 1) Load plan draft for this iteration
        // 2) Lookup current_knob in plan draft, determine malloc location
        entry = call_tuner (CUZMEM_TUNER_LOOKUP, &size);
        if (entry == NULL) {
            ret = hipErrorNotInitialized;
        } else {
            *devPtr = entry->gpu_pointer;
        }
    }

#if defined (DEBUG)
    printf ("libcuzmem: %s:%s | %i Bytes  [%i/%i]\n",
            project_name, plan_name, (unsigned int)(size),
            current_knob, num_knobs-1);
#endif

    // Morph CUDA Driver return codes into CUDA Runtime codes
    switch (ret)
    {
    case hipSuccess:
        return (hipSuccess);
    case hipErrorDeinitialized:
    case hipErrorNotInitialized:
    case hipErrorInvalidContext:
    case hipErrorInvalidValue:
    case hipErrorOutOfMemory:
    default:
        return (hipErrorOutOfMemory);
    }

}

hipError_t
hipFree (void *devPtr)
{
    hipError_t ret;
    cuzmem_plan *entry = NULL;

    // Decide how to free this chunk of gpu mapped memory
//    if (CUZMEM_RUN == op_mode) {
        entry = plan;

        // Lookup plan entry for this gpu pointer
        while (1) {
            if (entry == NULL) {
                fprintf (stderr, "libcuzmem: attempt to free invalid pointer (%p).\n", devPtr);
                exit (1);
            }
            if (entry->gpu_pointer == devPtr) {
                break;
            }
            entry = entry->next;
        }

        // Was it pinned cpu memory or real gpu memory?
        if (entry->cpu_pointer == NULL) {
            // real gpu memory
#if defined (DEBUG)
            printf ("libcuzmem: freeing %i\n", entry->id);
#endif
            ret = hipFree (entry->gpu_dptr);
            entry->gpu_pointer = NULL;
        } else {
            // pinned cpu memory
            // NOT YET IMPLEMENTED !
        }
//    }
//    else if (CUZMEM_TUNE == op_mode) {
        // NOT YET IMPLEMENTED !
//    }

    if (ret != hipSuccess) {
        fprintf (stderr, "libcuzmem: hipFree() failed\n");
    } 

    // Morph CUDA Driver return codes into CUDA Runtime codes
    switch (ret)
    {
    case hipSuccess:
        return (hipSuccess);
    case hipErrorDeinitialized:
    case hipErrorNotInitialized:
        return (hipErrorNotInitialized);
    case hipErrorInvalidContext:
    case hipErrorInvalidValue:
    case hipErrorOutOfMemory:
    default:
        return (hipErrorInvalidDevicePointer);
    }
}



//------------------------------------------------------------------------------
// CUDA RUNTIME REPLACEMENT HELPERS
//------------------------------------------------------------------------------

hipError_t
alloc_mem (cuzmem_plan* entry, size_t size)
{
    hipError_t ret;
    hipDeviceptr_t dev_mem;
    void* host_mem = NULL;

    if (entry->loc == 1) {
        // allocate gpu global memory
        ret = hipMalloc (&dev_mem, (unsigned int)size);

        // record in entry entry for hipFree() later on
        if (ret == hipSuccess) {
            entry->gpu_pointer = (void *)dev_mem;
            entry->gpu_dptr = dev_mem;
        }
    }
    else if (entry->loc == 0) {
        // allocate pinned host memory (probably broken for now)
        ret = hipMemAllocHost ((void **)&host_mem, (unsigned int)size);
        if (ret != hipSuccess) { return hipErrorInvalidValue; };
        ret = hipHostGetDevicePointer (&dev_mem, host_mem, 0);

        // record in entry entry for hipFree() later on
        if (ret == hipSuccess) {
            entry->cpu_pointer = (void *)host_mem;
            entry->gpu_pointer = (void *)dev_mem;
            entry->gpu_dptr = dev_mem;
        }
    }
    else {
        // unspecified memory location
        fprintf (stderr, "libcuzmem: entry specifed malloc to neither pinned nor global memory!\n");
        exit (1);
    }

    return ret;
}



//------------------------------------------------------------------------------
// FRAMEWORK FUNCTIONS
//------------------------------------------------------------------------------

// Called at start of each plan invocation
void
cuzmem_start (enum cuzmem_op_mode m)
{
#if defined (DEBUG)
    char debug_mode[20];
#endif

    // This state info is modified for all engines.
    current_knob = 0;
    op_mode = m;

#if defined (DEBUG)
    if (CUZMEM_RUN == op_mode) { strcpy (debug_mode, "CUZMEM_RUN"); }
    else if (CUZMEM_TUNE == op_mode) { strcpy (debug_mode, "CUZMEM_TUNE"); }
    else { printf ("libcuzmem: unknown operation mode specified! (exiting)\n"); exit (1); }
    printf ("libcuzmem: mode is %s\n", debug_mode);
#endif

    if (CUZMEM_RUN == op_mode) {
        plan = read_plan (project_name, plan_name);
    }
    // Invoke Tuner's "Start of Plan" routine.
    else if (CUZMEM_TUNE == op_mode) {
        call_tuner (CUZMEM_TUNER_START, NULL);
    }
    else {
        fprintf (stderr, "libcuzmem: unknown operation mode specified!\n");
    }
}


// Called at end of each plan invocation
cuzmem_op_mode
cuzmem_end ()
{
    // Ask the selected Tuner Engine what to do.
    if (CUZMEM_TUNE == op_mode) {
        call_tuner (CUZMEM_TUNER_END, NULL);
        tune_iter++;
    }

    // Return this back to calling program so that the
    // framework will know what to do next: next iteration
    // or stop iterating.
    return op_mode;
}



//------------------------------------------------------------------------------
// USER INTERFACE FUNCTIONS
//------------------------------------------------------------------------------

// Used to select/define CUZMEM project
void
cuzmem_set_project (char* project)
{
#if defined (DEBUG)
    fprintf (stderr, "libcuzmem: cuzmem_set_project() called\n");
#endif

    strcpy (project_name, project);
}


// Used to select/define CUZMEM plan
void
cuzmem_set_plan (char* plan)
{
#if defined (DEBUG)
    fprintf (stderr, "libcuzmem: cuzmem_set_plan() called\n");
#endif

    strcpy (plan_name, plan);
}


// Used to select Tuning Engine
void
cuzmem_set_tuner (enum cuzmem_tuner t)
{
    switch (t)
    {
    case CUZMEM_EXHAUSTIVE:
    default:
        call_tuner = cuzmem_tuner_exhaustive;
    }
}



//------------------------------------------------------------------------------
// TUNING ENGINES
//------------------------------------------------------------------------------

// The default Tuning Engine
cuzmem_plan*
cuzmem_tuner_exhaustive (enum cuzmem_tuner_action action, void* parm)
{
    if (CUZMEM_TUNER_START == action) {
        // For now, do nothing special.
        printf ("libcuzmem: iteration %i/%i\n", tune_iter, tune_iter_max);

        if (tune_iter == 0) {
            // if we are in the 0th tuning cycle, do nothing here.
            // CUZMEM_TUNER_LOOKUP is building a base plan draft and
            // is also determining the search space.
            return NULL;
        } else {
            // we now know the search space and we also know
            // that everything doesn't fit into GPU global

        }

        // Return value currently has no meaning
        return NULL;
    }
    else if (CUZMEM_TUNER_LOOKUP == action) {
        // parm: pointer to size of allocation
        size_t size = *(size_t*)(parm);

        hipError_t ret;
        bool is_inloop = false;
        cuzmem_plan* entry = NULL;

        // For the 0th iteration, build a base plan draft that
        // first fills GPU global memory and then spills over
        // into pinned CPU memory.
        if (tune_iter == 0) {
            // 1st try to detect if this allocation is an inloop entry.
            entry = plan;
            while (entry != NULL) {
                if ((entry->size == size) && (entry->gpu_pointer == NULL)) {
                    is_inloop = true;
                    break;
                }
                entry = entry->next;
            }

            if (is_inloop) {
                entry->inloop = 1;
                ret = alloc_mem (entry, size);
                if (ret != hipSuccess) {
                    // Note, hipMalloc() will report a NULL return value
                    // from call_tuner(LOOKUP) as hipErrorOutOfMemory
                    entry = NULL;
                }
            } else {
                entry = (cuzmem_plan*) malloc (sizeof(cuzmem_plan));
                entry->id = current_knob;
                entry->size = size;
                entry->loc = 1;
                entry->inloop = 0;
                entry->cpu_pointer = NULL;
                entry->gpu_pointer = NULL;

                ret = alloc_mem (entry, size);
                if (ret != hipSuccess) {
                    // out of gpu global memory: move to pinned CPU
                    entry->loc = 0;
                    ret = alloc_mem (entry, size);
                    if (ret != hipSuccess) {
                        // not enough CPU memory: return failure
                        free (entry);
                        entry = NULL;
                    }
                }

                // Insert successful entry into plan draft
                entry->next = plan;
                plan = entry;

                current_knob++;
            }
        } else {
            // tuning iteration is greater than zero

        }

        return entry;
    }
    else if (CUZMEM_TUNER_END == action) {
        cuzmem_plan* entry = NULL;
        bool all_global = true;

        // do special stuff @ end of tune iteration zero
        if (tune_iter == 0) {

            // check all entries for pinned host memory usage
            entry = plan;
            while (entry != NULL) {
                if (entry->loc != 1) {
                    all_global = false;
                    break;
                }
                entry = entry->next;
            }

            // quit now if everything fits in gpu memory
            if (all_global) {
                printf ("libcuzmem: auto-tuning complete.\n");
                op_mode = CUZMEM_RUN;
                write_plan (plan, project_name, plan_name);
                return NULL;
            }

            // if everything didn't fit, size up the search space
            num_knobs = current_knob;
            tune_iter_max = (unsigned int)pow (2, num_knobs);
        }

        // reset current knob for next tune iteration
        current_knob = 0;

        // TODO
        tune_iter = 99999999;

        // have we exhausted the search space?
        if (tune_iter >= tune_iter_max) {
            // if so, stop iterating
            printf ("libcuzmem: auto-tuning complete.\n");
            op_mode = CUZMEM_RUN;

            // ...and write out the plan
            write_plan (plan, "plastimatch", "foobaz");
        }

        // return value currently has no meaning
        return NULL;
    }
    else {
        printf ("libcuzmem: tuner asked to perform unknown action!\n");
        exit (1);
        return NULL;
    }
}
