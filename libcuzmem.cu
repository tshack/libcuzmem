/*  This file is part of libcuzptune
    Copyright (C) 2011  James A. Shackleford

    libcuzptune is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "libcuzmem.h"

#define DEBUG

char plan_name[FILENAME_MAX];
char project_name[FILENAME_MAX];
int (*search)() = cuzptune_search_exhaustive;
unsigned int current_knob = 0;
unsigned int num_knobs = 0;


hipError_t
hipMalloc (void **devPtr, size_t size)
{
    int use_global;
    hipError_t driver_return;
    hipDeviceptr_t dev_mem;
    void* host_mem;

    // decide what to do with this... knob
    use_global = search();

    if (use_global) {
        // allocate gpu global memory
        driver_return = hipMalloc (&dev_mem, (unsigned int)size);
    } else {
        // allocate pinned host memory (probably broken for now)
        driver_return = hipMemAllocHost ((void **)&host_mem, (unsigned int)size);
        if (driver_return != hipSuccess) { return hipErrorOutOfMemory; };
        driver_return = hipHostGetDevicePointer (&dev_mem, host_mem, 0);
    }
    *devPtr = (void *)dev_mem;

#if defined (DEBUG)
    printf ("*** | %s : %s | [%i KB] - #%i/%i\n",
            project_name, plan_name, (unsigned int)(size / 1024),
            current_knob, num_knobs-1);
#endif

    current_knob++;

    switch (driver_return)
    {
    case hipSuccess:
        return (hipSuccess);
    case hipErrorDeinitialized:
    case hipErrorNotInitialized:
    case hipErrorInvalidContext:
    case hipErrorInvalidValue:
    case hipErrorOutOfMemory:
    default:
        return (hipErrorOutOfMemory);
    }


}


int
cuzptune_search_exhaustive ()
{
    printf ("** Going with Global!\n");

    // use gpu global memory
    return 1;
}


void
cuzptune_search (enum cuzptune_search_mode mode)
{
    switch (mode)
    {
    case CUZPTUNE_EXHAUSTIVE:
    case CUZPTUNE_MAGIC:
    default:
        search = cuzptune_search_exhaustive;
    }
}


void
cuzptune_start ()
{
    current_knob = 0;
}


void
cuzptune_end ()
{
    if (current_knob > num_knobs) {
        num_knobs = current_knob;
    }
}


void
cuzptune_plan (char* plan)
{
    strcpy (plan_name, plan);
}


void
cuzptune_project (char* project)
{
    strcpy (project_name, project);
}

#if defined (commentout)
void
cuzptune_plan (int count, ...)
{
    int i;
    va_list knobs;

    va_start (knobs, count);

    for (i=0; i<count; i++) {
        
    }

    va_end (knobs);
}
#endif
