#include "hip/hip_runtime.h"
/*  This file is part of libcuzmem
    Copyright (C) 2011  James A. Shackleford

    libcuzmem is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "libcuzmem.h"
#include "plans.h"

//#define DEBUG

//------------------------------------------------------------------------------
// STATE SYMBOLS                            ...I know!
//------------------------------------------------------------------------------
char plan_name[FILENAME_MAX];
char project_name[FILENAME_MAX];
int (*call_tuner)(enum cuzmem_tuner_action) = cuzmem_tuner_exhaustive;
unsigned int current_knob = 0;
unsigned int num_knobs = 0;
unsigned int tune_iter = 0;
unsigned int tune_iter_max = 0;
enum cuzmem_op_mode op_mode = CUZMEM_RUN;

cuzmem_plan *plan = NULL;

//------------------------------------------------------------------------------
// CUDA RUNTIME REPLACEMENTS
//------------------------------------------------------------------------------

hipError_t
alloc_mem (cuzmem_plan* entry, size_t size)
{
    hipError_t ret;
    hipDeviceptr_t dev_mem;
    void* host_mem = NULL;

    if (entry->loc == 1) {
        // allocate gpu global memory
        ret = hipMalloc (&dev_mem, (unsigned int)size);

        // record in entry entry for hipFree() later on
        entry->gpu_pointer = (void *)dev_mem;
        entry->gpu_dptr = dev_mem;
    }
    else if (entry->loc == 0) {
        // allocate pinned host memory (probably broken for now)
        ret = hipMemAllocHost ((void **)&host_mem, (unsigned int)size);
        if (ret != hipSuccess) { return hipErrorInvalidValue; };
        ret = hipHostGetDevicePointer (&dev_mem, host_mem, 0);

        // record in entry entry for hipFree() later on
        entry->cpu_pointer = (void *)host_mem;
        entry->gpu_pointer = (void *)dev_mem;
        entry->gpu_dptr = dev_mem;
    }
    else {
        // unspecified memory location
        fprintf (stderr, "libcuzmem: entry specifed malloc to neither pinned nor global memory!\n");
        exit (1);
    }

    return ret;
}


hipError_t
hipMalloc (void **devPtr, size_t size)
{
    hipError_t ret;
    int use_global;
    cuzmem_plan *entry = NULL;

    *devPtr = NULL;

    // Decide what to do with current knob
    if (CUZMEM_RUN == op_mode) {
        // 1) Load plan for this project
        entry = plan;

        // 2) Lookup malloc type for this knob & allocate
        while (entry != NULL) {
            if (entry->id == current_knob) {
                ret = alloc_mem (entry, size);
                *devPtr = entry->gpu_pointer;
                break;
            }
            entry = entry->next;
        }

        // Knob id exceeds those found in plan... must be in a malloc/free loop
        if (*devPtr == NULL) {
#if defined (DEBUG)
            fprintf (stderr, "libcuzmem: malloc/free loop detected\n");
#endif

            // Look for a free()ed "inloop" marked plan entry 
            entry = plan;
            while (1) {
                if (entry == NULL) {
                    fprintf (stderr, "libcuzmem: unable to deduce allocation from plan!\n");
                    exit (1);
                }
                if ((entry->inloop == 1)         &&
                    (entry->gpu_pointer == NULL) &&
                    (entry->size == size)) {
#if defined (DEBUG)
                        printf ("libcuzmem: looking for %i byte plan entry ...found (%i).\n", (int)entry->size, entry->id);
#endif
                        ret = alloc_mem (entry, size);
                        if (ret != hipSuccess) {
                            fprintf (stderr, "libcuzmem: inloop alloc_mem() failed [%i]\n", ret);
                        }
                        *devPtr = entry->gpu_pointer;
                        break;
                }
                entry = entry->next;
            }
        } else {
            current_knob++;
        }
    }
    else if (CUZMEM_TUNE == op_mode) {
        // 1) Load plan draft for this iteration

        // 2) Lookup current_knob in plan draft, determine malloc location
        use_global = call_tuner (CUZMEM_TUNER_LOOKUP);

        // 3) Allocate either pinned host or global device memory

        // Get ready for next knob
        current_knob++;
    }

#if defined (DEBUG)
    printf ("libcuzmem: %s:%s | %i Bytes  [%i/%i] ondev:%i\n",
            project_name, plan_name, (unsigned int)(size),
            current_knob, num_knobs-1, use_global);
#endif

    // Morph CUDA Driver return codes into CUDA Runtime codes
    switch (ret)
    {
    case hipSuccess:
        return (hipSuccess);
    case hipErrorDeinitialized:
    case hipErrorNotInitialized:
    case hipErrorInvalidContext:
    case hipErrorInvalidValue:
    case hipErrorOutOfMemory:
    default:
        return (hipErrorOutOfMemory);
    }


}

hipError_t
hipFree (void *devPtr)
{
    hipError_t ret;
    cuzmem_plan *entry = NULL;

    // Decide how to free this chunk of gpu mapped memory
    if (CUZMEM_RUN == op_mode) {
        entry = plan;

        // Lookup plan entry for this gpu pointer
        while (1) {
            if (entry == NULL) {
                fprintf (stderr, "libcuzmem: attempt to free invalid pointer (%p).\n", devPtr);
                exit (1);
            }
            if (entry->gpu_pointer == devPtr) {
                break;
            }
            entry = entry->next;
        }

        // Was it pinned cpu memory or real gpu memory?
        if (entry->cpu_pointer == NULL) {
            // real gpu memory
#if defined (DEBUG)
            printf ("libcuzmem: freeing %i\n", entry->id);
#endif
            ret = hipFree (entry->gpu_dptr);
            entry->gpu_pointer = NULL;
        } else {
            // pinned cpu memory
            // NOT YET IMPLEMENTED !
        }
    }
    else if (CUZMEM_TUNE == op_mode) {
        // NOT YET IMPLEMENTED !
    }

    if (ret != hipSuccess) {
        fprintf (stderr, "libcuzmem: hipFree() failed\n");
    } 

    // Morph CUDA Driver return codes into CUDA Runtime codes
    switch (ret)
    {
    case hipSuccess:
        return (hipSuccess);
    case hipErrorDeinitialized:
    case hipErrorNotInitialized:
        return (hipErrorNotInitialized);
    case hipErrorInvalidContext:
    case hipErrorInvalidValue:
    case hipErrorOutOfMemory:
    default:
        return (hipErrorInvalidDevicePointer);
    }
}

//------------------------------------------------------------------------------
// FRAMEWORK FUNCTIONS
//------------------------------------------------------------------------------

// Called at start of each plan invocation
void
cuzmem_start (enum cuzmem_op_mode m)
{
    char debug_mode[20];

    // This state info is modified for all engines.
    current_knob = 0;
    op_mode = m;

#if defined (DEBUG)
    if (CUZMEM_RUN == op_mode) {
        strcpy (debug_mode, "CUZMEM_RUN");
    }
    else if (CUZMEM_TUNE == op_mode) {
        strcpy (debug_mode, "CUZMEM_TUNE");
    }
    else {
        printf ("libcuzmem: unknown operation mode specified! (exiting)\n");
        exit (1);
    }

    printf ("libcuzmem: mode is %s\n", debug_mode);
#endif

    plan = read_plan (project_name, plan_name);

    // Invoke Tuner's "Start of Plan" routine.
    call_tuner (CUZMEM_TUNER_START);
}


// Called at end of each plan invocation
cuzmem_op_mode
cuzmem_end ()
{
    // Ask the selected Tuner Engine what to do.
    call_tuner (CUZMEM_TUNER_END);

    // Return this back to calling program so that the
    // framework will know what to do next: next iteration
    // or stop iterating.
    return op_mode;
}



//------------------------------------------------------------------------------
// USER INTERFACE FUNCTIONS
//------------------------------------------------------------------------------

// Used to select/define CUZMEM project
void
cuzmem_set_project (char* project)
{
    strcpy (project_name, project);
}


// Used to select/define CUZMEM plan
void
cuzmem_set_plan (char* plan)
{
    strcpy (plan_name, plan);
}


// Used to select Tuning Engine
void
cuzmem_set_tuner (enum cuzmem_tuner t)
{
    switch (t)
    {
    case CUZMEM_EXHAUSTIVE:
    default:
        call_tuner = cuzmem_tuner_exhaustive;
    }
}



//------------------------------------------------------------------------------
// TUNING ENGINES
//------------------------------------------------------------------------------

// The default Tuning Engine
int
cuzmem_tuner_exhaustive (enum cuzmem_tuner_action action)
{
    if (CUZMEM_RUN == op_mode) {
        if (CUZMEM_TUNER_START == action) {
            // For now, do nothing special.
            return 0;
        }
        else if (CUZMEM_TUNER_LOOKUP == action) {
            // For now, just blindly put everything in device global
            return 1;
        }
        else if (CUZMEM_TUNER_END == action) {
            current_knob = 0;
            return 0;
        }
        else {
            printf ("libcuzmem: tuner asked to perform unknown run action!\n");
            exit (1);
        }
    }
    else if (CUZMEM_TUNE == op_mode) {
        if (CUZMEM_TUNER_START == action) {
            // For now, do nothing special.
            printf ("libcuzmem: iteration %i/%i\n", tune_iter, tune_iter_max);
            return 0;
        }
        else if (CUZMEM_TUNER_LOOKUP == action) {
            // For now, just blindly put everything in device global
            return 1;
        }
        else if (CUZMEM_TUNER_END == action) {
            // Record # of malloc encounters
            if (current_knob > num_knobs) {
                num_knobs = current_knob;
                tune_iter_max = (unsigned int)pow (2, num_knobs);
            }

            // Reset current knob for next iteration
            current_knob = 0;

            // Increment tune iteration count
            tune_iter++;

            // Have we exhausted the search space?
            if (tune_iter >= tune_iter_max) {
                // If so, stop iterating
                op_mode = CUZMEM_RUN;
            }
                return 0;
            }
        else {
            printf ("libcuzmem: tuner asked to perform unknown tune action!\n");
            exit (1);
        }
    }

    // We should never get to here
    exit (1);
    return 0;
}
